#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include<iostream>
#include<vector>

#define NUM_THREADS 256



struct Bin{
    particle_t** particles;
    int number_of_particles;
};




extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}


//original code
/* 
__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}
*/



__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}






__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}


//method called by host to send the grid to gpu
__host__ Bin* send_grid_to_gpu(Bin* grid, int dim){
    Bin* d_grid;
    hipMalloc((void **) &d_grid, dim * dim * sizeof(Bin));
    hipMemcpy(d_grid, grid, dim * dim * sizeof(Bin), hipMemcpyHostToDevice);
    return d_grid;
}


__device__ void apply_forces_to_cell(Bin &src, Bin &neighbor){
    int num_particles_src = src.number_of_particles;
    int num_particles_neighbor = neighbor.number_of_particles;

    for(int i = 0; i < num_particles_src; i++){
        for(int j = 0; j < num_particles_neighbor; j++){
            apply_force_gpu(*(src.particles[i]), *(neighbor.particles[j]));
        }
    }

}

//method to apply the forces on the bins
__device__ void apply_forces_on_grid(Bin* grid, const int dim, int tid){

    // assume that # of threads are <dim>
    int bin_i = tid/dim;
    int bin_j = tid%dim;
    int index = bin_i * dim + bin_j;
    int number_of_particles = grid[index].number_of_particles;
    
    // initilize acceleration
    for(int i = 0; i < number_of_particles; i++){
        grid[index].particles[i]->ax = 0;
        grid[index].particles[i]->ay = 0;        
    }
    
    for(int i = -1; i < 2; i++){
        int delta_i = bin_i + i;
        for(int j = -1; j < 2; j++){
            int delta_j = bin_j + j;
            if(delta_i >= 0 && delta_j >= 0 && delta_i < dim && delta_j < dim){
                int index2 = delta_i * dim + delta_j;
                apply_forces_to_cell(grid[index], grid[index2]);
            }
        }
    }
   
} 



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );


    std::cout << "size: " << size << std::endl;
    double world_dim = size;
    int grid_dim = int(ceil(sqrt(n)));
    double cell_size = world_dim/grid_dim;
    //ensure to not violate cutoff constraint
    // if violated, set cell_size to be the cutoff
    std::cout<< "cutoff: " << cutoff << std::endl; 
    if(cell_size < cutoff){
        grid_dim = int(world_dim / cutoff);
        cell_size= world_dim / grid_dim; 
    }

    //init grid
    Bin* grid = (Bin*) malloc(grid_dim*grid_dim*sizeof(Bin));
    for(int i = 0; i < grid_dim*grid_dim; i++)
        grid[i] = Bin(); 

    //used vector in host temporarily to bin particles in host
    // then papck the particles in the grid array to be used in gpu
    std::vector<particle_t*> temp[grid_dim][grid_dim];
    for(int i = 0; i < n; i++){
        int index_i = floor(particles[i].y/cell_size);
        int index_j = floor(particles[i].x/cell_size);
        temp[index_i][index_j].push_back(&particles[i]);
    }
    //naive way,, improve later
    for(int i = 0; i < grid_dim; i++){
        for(int j = 0; j < grid_dim; j++){
            int number_of_particles = temp[i][j].size();
            grid[i*grid_dim + j].number_of_particles = number_of_particles;
            grid[i*grid_dim + j].particles = (particle_t**) malloc(number_of_particles * sizeof(particle_t*));
            for(int k = 0; k < number_of_particles; k++){
                grid[i*grid_dim + j].particles[k] = temp[i][j][k];
            }
            temp[i][j].clear();     
        }
    }

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

	int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
        
        //
        //  move particles
        //
	move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
